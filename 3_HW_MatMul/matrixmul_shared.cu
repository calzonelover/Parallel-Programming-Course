#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "matrixmul.h"

#define BLOCK_SIZE 32
__global__ void matmu(float *_A, float *_B, float *_C);


int main(){
	float *hA, *hB, *hC;
	// time
	hipEvent_t start, stop;
  	float gpu_time;
  	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 B(BLOCK_SIZE, BLOCK_SIZE);
	dim3 G(nx/BLOCK_SIZE+1, ny/BLOCK_SIZE+1);
	// host
	hA = (float*)malloc(WA*HA*sizeof(float));
	hB = (float*)malloc(WB*HB*sizeof(float));
	hC = (float*)malloc(WB*HA*sizeof(float));
	srand(time(NULL));
	initMatrix(hA, WA, HA);
	initMatrix(hB, WB, HB);
	// device
	hipMalloc((void **)&A, size);
	hipMalloc((void **)&B, size);
	hipMalloc((void **)&C, size);
	hipMemcpy(A, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(B, hB, size, hipMemcpyHostToDevice);
	hipMemcpy(C, hC, size, hipMemcpyHostToDevice);
	// compute and record
	hipEventRecord(start, 0);
	matmul<<<G, B>>>(A, B, C);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("GPU time = %f s\n", gpu_time*0.001);
	hipMemcpy(A, hA, size, hipMemcpyDeviceToHost);
	hipMemcpy(B, hB, size, hipMemcpyDeviceToHost);
	hipMemcpy(C, hC, size, hipMemcpyDeviceToHost);
	// printf("Matrix A =\n");
	// printMatrix(A, WA, HA);
	// printf("Matrix B =\n");
	// printMatrix(B, WB, HB);
	// printf("Matrix C =\n");
	// printMatrix(C, WB, HA);
	hipFree(A);
	hipFree(B);
	hipFree(C);
	free(hA);
	free(hB);
	free(hC);
	return 0;
}

void initMatrix(float *_M, int _W, int _H){
	for (unsigned int h=0; h<_H;h++){
		for (unsigned int w=0; w<_W; w++){
			_M[w+h*WA] = (float)rand()/ (float)RAND_MAX;//(int)rand() % 16;
		}
	}
}

void printMatrix(float *_M, int _W, int _H){
	for (unsigned int h=0; h<_H;h++){
		printf("%d|\t", h);
		for (unsigned int w=0; w<_W; w++){
			printf("%f\t", _M[w+h*_W]);
		}
		printf("|\n");
	}
}

__global__ void matmu(float *_A, float *_B, float *_C){
	__shared__ float sumover;
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;
	int indx = ix+iy*nx;
	if (ix > 0 && iy > 0 && i < HA && j < WB){
		float sumover = 0.0f;
		for (unsigned int dmmy = 0; dmmy< HB; dmmy++){
			sumover += _A[dmmy+i*WA]*_B[j+dmmy*WB];
		}
		_C[j+i*WB] = sumover;
	}
}