#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "matrixmul.h"

#define BLOCK_SIZE 32
#define STRIDE_SIZE 32

__global__ void matmul(float *_A, float *_B, float *_C);

int main(){
	float *hA, *hB, *hC;
	// time
	hipEvent_t start, stop;
  	float gpu_time;
  	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(WB/BLOCK_SIZE+1, HA/BLOCK_SIZE+1);
	// allocate mem
	hA = (float*)malloc(WA*HA*sizeof(float));
	hB = (float*)malloc(WB*HB*sizeof(float));
	hC = (float*)malloc(WB*HA*sizeof(float));
	hipMalloc((void **)&A, WA*HA*sizeof(float));
	hipMalloc((void **)&B, WB*HB*sizeof(float));
	hipMalloc((void **)&C, WB*HA*sizeof(float));
	// init value
	initMatrix(hA, WA, HA);
	initMatrix(hB, WB, HB);
	hipMemcpy(A, hA, WA*HA*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B, hB, WB*HB*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C, hC, WB*HA*sizeof(float), hipMemcpyHostToDevice);
	// check if there is no matching dim of matrix mul
	if (WA != HB){
		hipFree(A);
		hipFree(B);
		hipFree(C);
		free(hA);
		free(hB);
		free(hC);
		exit(0);
		return 0;
	}
	hipEventRecord(start);
	// compute and record
	for (unsigned int iter = 0; iter < MAX_ITER ; iter++)
		matmul<<<dimGrid, dimBlock>>>(A, B, C);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("GPU time = %f s\n", gpu_time*0.001/MAX_ITER);
	hipMemcpy(hA, A, WA*HA*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hB, B, WB*HB*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hC, C, WB*HA*sizeof(float), hipMemcpyDeviceToHost);
	// printf("Matrix A =\n");
	// printMatrix(hA, WA, HA);
	// printf("Matrix B =\n");
	// printMatrix(hB, WB, HB);
	// printf("Matrix C =\n");
	// printMatrix(hC, WB, HA);
	hipFree(A);
	hipFree(B);
	hipFree(C);
	free(hA);
	free(hB);
	free(hC);
	return 0;
}

void initMatrix(float *_M, int _W, int _H){
	srand(time(NULL));
	for (unsigned int h=0; h<_H;h++){
		for (unsigned int w=0; w<_W; w++){
			_M[w+h*_W] = (int)rand() % 16;//(float)rand()/ (float)RAND_MAX;
		}
	}
}

void printMatrix(float *_M, int _W, int _H){
	for (unsigned int h=0; h<_H;h++){
		printf("%d|\t", h);
		for (unsigned int w=0; w<_W; w++){
			printf("%f\t", _M[w+h*_W]);
		}
		printf("|\n");
	}
}


__global__ void matmul(float *_A, float *_B, float *_C){
	__shared__ float _Asub[STRIDE_SIZE][STRIDE_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < WB && j < HA){
		_Asub[threadIdx.y][threadIdx.x] = _A[j*STRIDE_SIZE+threadIdx.x];
		float sumoverpad = 0.0f;
		for (unsigned int dmmy = 0; dmmy < STRIDE_SIZE; dmmy++){
			sumoverpad += _Asub[threadIdx.y][dmmy]*_B[i+dmmy*WB];
		}
		_C[i+j*WB] = sumoverpad;
	}
}