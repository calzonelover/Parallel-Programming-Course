#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

__global__ void reduce(int *g_idata, int *g_odata){
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x; 
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();
  
  for(unsigned int s = blockDim.x/2;s>0;s>>=1){
    if(tid<s){
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();
  }
  
  if(tid==0) g_odata[blockIdx.x] = sdata[0];
}

///////////////////////////////////////////////////////////////////////////

void init(int *A, int nA) {
  for (int i=0; i<nA; i++){
      A[i] = (int)rand() % 16;
  }
}

int c_summation(int *A, int nA) {
  int sum=A[0];
  for (int i=1; i<nA; i++) {
        sum += A[i];
  }
  return sum;
}

int main() {
  int order = 18;
  int nA = pow(2,order);
  printf("Vector size: %d\n", nA); 
  size_t sizeA = nA*sizeof(int);
  int c_sum,pr_sum;
  int *A,*S;
  A = (int*) malloc(sizeA); 
  S = (int*) malloc(sizeA); 
  
  srand(time(NULL));
  init(A, nA);
  
  int *dA =NULL;
  int *dS =NULL;
  hipMalloc((void**)&dA,sizeA);
  hipMalloc((void**)&dS,sizeA);
  hipMemcpy(dA,A,sizeA,hipMemcpyHostToDevice);
  //cudaMemcpy(dS,S,sizeA,cudaMemcpyHostToDevice);

  int B = 1024;
  int G = (nA+B-1)/B;
  int smemSize = B*sizeof(int);
  
  float kernel_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  //////////////////////////// Start time record
  reduce<<<G,B,smemSize>>>(dA,dS);
  // for (unsigned int stride=B; stride < 1 ; stride >>= 1){
  for(int problemsize=nA/B;problemsize > 1;problemsize/=B){
    reduce<<<G,B,smemSize>>>(dS,dS);
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&kernel_time,start,stop);
  printf("GPU time = %lf ms\n", kernel_time);
  
  hipMemcpy(S,dS,sizeA,hipMemcpyDeviceToHost);
  pr_sum = S[0]; 
  
  clock_t begin, end;
  begin = clock();
  c_sum = c_summation(A,nA);
  end = clock();
  double cpu_time = (double) (end-begin)/ CLOCKS_PER_SEC *1000.0f;
  printf("CPU time = %lf ms\n", cpu_time);
  
  printf("Parallel reduction sum result is: %d \n",pr_sum);
  printf("c function sum result is: %d \n", c_sum);

  free(A);
  hipFree(dA);
  
  return 0;
}
