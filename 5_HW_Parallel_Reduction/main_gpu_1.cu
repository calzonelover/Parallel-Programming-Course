#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main.h"


#define BLOCK_SIZE 128


void arrValidation(int _real_sum, int _sum);
// bool checkBaseValue(int _N);
void initValue(int *_arr);
int cpuSumArray(int *_arr);

template <class T>
__global__ void sumArray(T *_arr, T *_oarr);



int main (int argc, char *argv[]) {
	float gpu_time;
	int true_sum;
	int *d_arr, *d_oarr;
	hipEvent_t start_t, stop_t;
	int blocks = ARR_SIZE/BLOCK_SIZE;
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(blocks, 1, 1);
	size_t sizeArr = ARR_SIZE*sizeof(int);
	size_t smSize = BLOCK_SIZE*sizeof(int);

	hipEventCreate(&start_t);
	hipEventCreate(&stop_t);
	arr = (int*)malloc(sizeArr);
	hipMalloc((void **)&d_arr, sizeArr);
	hipMalloc((void **)&d_oarr, sizeArr);

	srand(time(NULL));
	initValue(arr);

	true_sum = cpuSumArray(arr);

	hipMemcpy(d_arr, arr, sizeArr, hipMemcpyHostToDevice);
	int workamount = (blocks > 1) ? ARR_SIZE/BLOCK_SIZE + 1 : 0;
	hipEventRecord(start_t);
	sumArray<<<dimGrid, dimBlock, smSize>>>(d_arr, d_oarr);
	for (unsigned int dmmy=0; dmmy < workamount; dmmy++){
		sumArray<<<dimGrid, dimBlock, smSize>>>(d_oarr, d_oarr);
	}
	hipEventRecord(stop_t);
	hipEventSynchronize(stop_t);
	hipEventElapsedTime(&gpu_time, start_t, stop_t);
	printf("GPU time = %lf ms\n", gpu_time/ITER_TIMES);

	hipMemcpy(arr, d_oarr, sizeArr, hipMemcpyDeviceToHost);
	arrValidation(true_sum, arr[0]);

	free(arr);
	hipFree(d_arr);
    return 0;
}







void initValue(int *_arr){
	for(unsigned int i=0; i<ARR_SIZE; i++){
		_arr[i] = (int)rand() % 9;
	}
}

void arrValidation(int _real_sum, int _sum){
	std::cout << _real_sum << std::endl;
	std::cout << _sum << std::endl;
	std::cout << _real_sum - _sum << std::endl;
	if (_real_sum == _sum){
		std::cout << "array summation is right !\n" << std::endl;
	} else{
		std::cout << "array summation is wrong !\n" << std::endl;
	}
}

int cpuSumArray(int *_arr){
	int sum = _arr[0];
	for(unsigned int i=1; i<ARR_SIZE; i++){
		sum += _arr[i];
	}
	return sum;
}


template <class T>
__global__ void sumArray(T *_arr, T *_oarr){
	__shared__ T _sarr[BLOCK_SIZE];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	_sarr[tid] = (i < ARR_SIZE) ? _arr[i]+_arr[i+blockDim.x] : 0;
	__syncthreads();
    for (unsigned int stride = blockDim.x/2; stride > 0 ; stride >>= 1){
	if (tid < stride)
		_sarr[tid]+=_sarr[tid+stride];
	__syncthreads();
    }
	if (tid == 0) _oarr[blockIdx.x] = _sarr[0];
}
