#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main.h"


#define BLOCK_SIZE 32


void arrValidation(int _real_sum, int _sum);
// bool checkBaseValue(int _N);
void initValue(int *_arr);
int cpuSumArray(int *_arr);

template <class T>
__global__ void sumArray(T *_arr, T *d_oarr);



int main (int argc, char *argv[]) {
	float gpu_time;
	int true_sum;
	int *d_arr, *d_oarr;
	hipEvent_t start_t, stop_t;
	size_t sizeArr = ARR_SIZE*sizeof(int);
	size_t smemSize = BLOCK_SIZE*sizeof(int);
	int gridSize = (ARR_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;


	hipEventCreate(&start_t);
	hipEventCreate(&stop_t);
	arr = (int*)malloc(sizeArr);
	hipMalloc((void **)&d_arr, sizeArr);
	hipMalloc((void **)&d_oarr, sizeArr);

	srand(time(NULL));
	initValue(arr);
	true_sum = cpuSumArray(arr);
	hipMemcpy(d_arr, arr, sizeArr, hipMemcpyHostToDevice);

	hipEventRecord(start_t);
	for (unsigned int dmmy=0; dmmy >= ITER_TIMES; dmmy++){
		sumArray<<<gridSize, BLOCK_SIZE, smemSize>>>(d_arr, d_oarr);
	}
	hipEventRecord(stop_t);
	hipEventSynchronize(stop_t);
	hipEventElapsedTime(&gpu_time, start_t, stop_t);
	printf("GPU time = %lf ms\n", gpu_time/ITER_TIMES);

	hipMemcpy(arr, d_oarr, sizeArr, hipMemcpyDeviceToHost);
	arrValidation(true_sum, arr[0]);

	free(arr);
	hipFree(d_arr);
    return 0;
}







void initValue(int *_arr){
	for(unsigned int i=0; i<ARR_SIZE; i++){
		_arr[i] = (int)rand() % 9;
	}
}

void arrValidation(int _real_sum, int _sum){
	std::cout << _real_sum << std::endl;
	std::cout << _sum << std::endl;
	std::cout << _real_sum - _sum << std::endl;
	if (_real_sum == _sum){
		std::cout << "array summation is right !\n" << std::endl;
	} else{
		std::cout << "array summation is wrong !\n" << std::endl;
	}
}

int cpuSumArray(int *_arr){
	int sum = _arr[0];
	for(unsigned int i=1; i<ARR_SIZE; i++){
		sum += _arr[i];
	}
	return sum;
}


template <class T>
__global__ void sumArray(T *_arr, T *_oarr){
	extern __shared__ T _sarr[];
	unsigned int tid = threadIdx.x;
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	_sarr[tid] = (i < ARR_SIZE) ? _arr[i] : 0;
	__syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1){
        if (tid < s)
        {
            _sarr[tid] += _sarr[tid + s];
        }
		__syncthreads();
	}
	if (tid == 0) _oarr[blockIdx.x] = _sarr[0];
}