
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
__global__ void
MatAdd(const float *A, const float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < N && j < N)
    {
        int indx = i + j*N;
        C[indx] = A[indx] + B[indx];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Print the vector length to be used, and compute its size
    int N = 100;
    size_t size = N*N*sizeof(float);
    printf("[Matrix size of %dx%d elements]\n", N, N);
    // allocate memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N ; ++j){
            h_A[i+j*N] = 1;//rand()/(float)RAND_MAX;
            h_B[i+j*N] = 1; rand()/(float)RAND_MAX;
        }
    }
    // ALLOCATE DEVICE MEMORY
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    /// Beware no more than that 
    // Launch the Vector Add CUDA Kernel
    dim3 B(32,32);// B.x=32, B.y=32, B.z=1
    dim3 G(N/32 +1, N/32+1);
    printf("CUDA kernel launch with %dx%dx%d blocks of %dx%dx%d threads\n", B.x, B.y, B.z, G.x, G.y, G.z);
    MatAdd<<<G, B>>>(d_A, d_B, d_C, N);

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // verify the result
    float sum = 0;
    for (unsigned int i=0; i < N ; ++i){
        for (unsigned int j=0; j < N; ++j){
            sum += h_C[i+j*N];
        }
    }
    if (fabs(sum-N*N*2) < 1.0e-5){
        printf("Test PASSED\n");
    }else{
        printf("Test Don't pass: you fucked up!\n");
    }

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

