#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "wave2d.h"

#define BLOCK_SIZE 32

__global__ void fd(float *u0, float *u1, float *u2, float C2);

__global__ void update(float *u0, float *u1, float *u2);

int main(){
  hipEvent_t start, stop;
  float gpu_time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float *h_u0, *h_u1;
  float dx = xmax/(nx-1);
  C = v*dt/dx;
  C2 = C*C;
  size = nx*ny*sizeof(float);
  dim3 B(BLOCK_SIZE,BLOCK_SIZE);
  dim3 G(nx/BLOCK_SIZE+1, ny/BLOCK_SIZE+1);
  h_u0 = (float*) malloc(size);
  h_u1 = (float*) malloc(size);
  hipMalloc((void **)&u0, size);
  hipMalloc((void **)&u1, size);
  hipMalloc((void **)&u2, size);
  for (unsigned int iy=0; iy<ny; iy++){
    float y_now = iy*dx - 0.5*ymax;
    for (unsigned int ix=0; ix<nx; ix++){
        int indx = ix+iy*nx;
        float x_now = ix*dx - 0.5*xmax;
        h_u0[indx] = exp(-a*(pow(x_now,2)+pow(y_now,2)));
        h_u1[indx] = h_u0[indx];
    }
  }
  hipMemcpy(u0, h_u0, size, hipMemcpyHostToDevice);
  hipMemcpy(u1, h_u1, size, hipMemcpyHostToDevice);
  hipEventRecord(start, 0);
  for (unsigned int it_time=0; it_time<MAX_ITER; it_time++){
    for (unsigned int it=0; it<nt; it++) {
      fd<<<G,B>>>(u0, u1, u2, C2);
      update<<<G,B>>>(u0, u1, u2);
    }
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpu_time, start, stop);
  printf("GPU time = %f s\n", gpu_time*0.001/MAX_ITER);
  hipMemcpy(h_u0, u2, size, hipMemcpyDeviceToHost);
  FILE *file = fopen("uGPU_shared.dat","w");
  fwrite(h_u0, sizeof(float), nx*ny, file);
  fclose(file);
  //
  free(h_u0);
  free(h_u1);
  hipFree(u0);
  hipFree(u1);
  hipFree(u2);
  return 0;
}


__global__ void fd(float *u0, float *u1, float *u2, float C2){
  __shared__ float _u0[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float _u1[BLOCK_SIZE][BLOCK_SIZE];
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  int indx = ix+iy*nx;
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1){
    _u0[threadIdx.y][threadIdx.x] = u0[iy*BLOCK_SIZE+threadIdx.x];
    _u1[threadIdx.y][threadIdx.x] = u1[threadIdx.y*nx+ix];
  } else {
    _u0[threadIdx.y][threadIdx.x] = 0.0f;
    _u1[threadIdx.y][threadIdx.x] = 0.0f;
  }
  __syncthreads();
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1)
    u2[indx] = (2.0f-4.0f*C2)*_u1[ix][iy] - _u0[ix][iy]
              + C2*(_u1[ix+1][iy]+_u1[ix-1][iy]
              + _u1[ix][iy+1]+_u1[ix][iy-1]);
}


__global__ void update(float *u0, float *u1, float *u2){
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1){
   int indx = ix+iy*nx;
   u0[indx] = u1[indx];
   u1[indx] = u2[indx];
  }
}



