#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "wave2d.h"


__global__ void fd(float *u0, float *u1, float *u2, float C2){
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  int indx = ix+iy*nx;
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1)
    u2[indx] = (2.0f-4.0f*C2)*u1[indx] - u0[indx]
              + C2*(u1[(ix+1)+iy*nx]+u1[(ix-1)+iy*nx]
              + u1[ix+(iy+1)*nx]+u1[ix+(iy-1)*nx]);
    //u1[indx+1]+u1[indx-1]-u0[indx];
}

__global__ void update(float *u0, float *u1, float *u2){
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1){
   int indx = ix+iy*nx;
   u0[indx] = u1[indx];
   u1[indx] = u2[indx];
  }
}

int main(){
  hipEvent_t start, stop;
  float gpu_time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float *h_u0, *h_u1;
  float dx = xmax/(nx-1);
  C = v*dt/dx;
  C2 = C*C;
  size = nx*ny*sizeof(float);
  dim3 B(32,32);
  dim3 G(nx/32+1, ny/32+1);
  h_u0 = (float*) malloc(size);
  h_u1 = (float*) malloc(size);
  hipMalloc((void **)&u0, size);
  hipMalloc((void **)&u1, size);
  hipMalloc((void **)&u2, size);
  for (unsigned int iy=0; iy<ny; iy++){
    float y_now = iy*dx - 0.5*ymax;
    for (unsigned int ix=0; ix<nx; ix++){
        int indx = ix+iy*nx;
        float x_now = ix*dx - 0.5*xmax;
        h_u0[indx] = exp(-a*(pow(x_now,2)+pow(y_now,2)));
        h_u1[indx] = h_u0[indx];
    }
  }
  hipEventRecord(start, 0);
  hipMemcpy(u0, h_u0, size, hipMemcpyHostToDevice);
  hipMemcpy(u1, h_u1, size, hipMemcpyHostToDevice);
  for (unsigned int it=0; it<nt; it++) {
    fd<<<G,B>>>(u0, u1, u2, C2);
    update<<<G,B>>>(u0, u1, u2);
  }
  hipMemcpy(h_u0, u2, size, hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpu_time, start, stop);
  printf("GPU time = %f s\n", gpu_time*0.001);
  FILE *file = fopen("uGPU.dat","w");
  fwrite(h_u0, sizeof(float), nx*ny, file);
  fclose(file);
  //
  free(h_u0);
  free(h_u1);
  hipFree(u0);
  hipFree(u1);
  hipFree(u2);
  return 0;
}






