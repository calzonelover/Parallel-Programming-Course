#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "wave2d.h"

__global__ void fd(float *u0, float *u1, float *u2){
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  int indx = ix+iy*nx;
  if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1)
    u2[indx] = u1[indx+1]+u1[indx-1]-u0[indx];
}

__global__ void update(float *u0, float *u1, float *u2){
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iy = threadIdx.y + blockIdx.y*blockDim.y;
  int indx = ix+iy*nx;
  u0[indx] = u1[indx];
  u1[indx] = u2[indx];
}

int main(){
  hipEvent_t start, stop;
  float gpu_time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  size = nx*sizeof(float);
  float *h_u0, *h_u1;
  float dx = xmax/(nx-1);
  float dt = dx/v;
  h_u0 = (float*) malloc(size);
  h_u1 = (float*) malloc(size);
  hipMalloc((void **)&u0, size);
  hipMalloc((void **)&u1, size);
  hipMalloc((void **)&u2, size);
  for (unsigned int ix=0; ix<nx; ix++){
    float x = ix*dx;
    h_u0[ix] = exp(-a*pow(x-0.5*xmax,2.0));
    h_u1[ix] = exp(-a*pow(x-0.5*xmax-v*dt,2.0));
  }
  hipEventRecord(start, 0);
  hipMemcpy(u0, h_u0, size, hipMemcpyHostToDevice);
  hipMemcpy(u1, h_u1, size, hipMemcpyHostToDevice);
  for (unsigned int it=0; it<nt; it++) {
    fd<<<1,nx>>>(u0, u1, u2);
    update<<<1,nx>>>(u0, u1, u2);
  }
  hipMemcpy(h_u0, u2, size, hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpu_time, start, stop);
  printf("GPU time = %f s\n", gpu_time*0.001);
  FILE *file = fopen("uGPU.dat","w");
  fwrite(h_u0, sizeof(float), nx, file);
  fclose(file);
  return 0;
}






