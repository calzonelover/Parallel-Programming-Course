
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
__host__ void cpuFunction(){}
__device__ void gpuFunction(){}
__global__ void fd(float *u0, float *u1, float *u2, int nx) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  if (ix > 0 && ix < nx-1)
    u2[ix] = u1[ix+1]+u1[ix-1]-u0[ix];
}
__global__ void update(float *u0, float *u1, float *u2) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  u0[ix] = u1[ix];
  u1[ix] = u2[ix];
}
int main() {
  int ix, it, nx = 100, nt = 100;
  size_t size = nx*sizeof(float);
  float xmax = 1.0, dx = xmax/(nx+1);
  float *hU0, *hU1;
  float *u0, *u1, *u2, c = 0.1, dt = dx/c, x, a = 1000;
  hU0 = (float*) malloc(size);
  hU1 = (float*) malloc(size);
  hipMalloc((void **)&u0, size);
  hipMalloc((void **)&u1, size);
  hipMalloc((void **)&u2, size);
  for (ix=0; ix<nx; ix++) {
    x = ix*dx;
    hU0[ix] = exp(-a*pow(x-0.5*xmax,2.0));
    hU1[ix] = exp(-a*pow(x-0.5*xmax-c*dt,2.0));
  }
  hipMemcpy(u0, hU0, size, hipMemcpyHostToDevice);
  hipMemcpy(u1, hU1, size, hipMemcpyHostToDevice);
  for (it=0; it<nt; it++) {
    fd<<<1,nx>>>(u0, u1, u2, nx);
    update<<<1,nx>>>(u0, u1, u2);
  }
  hipMemcpy(hU0, u2, size, hipMemcpyDeviceToHost);
  FILE *file = fopen("u.dat","w");
  fwrite(hU0, sizeof(float), nx, file);
  fclose(file);
  return 0;
}
