#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "mandelbrot.h"

__global__ void 
MandelFunc (float *Zx, float *Zy, int *K
, float dx, float dy
, float x_i, float y_i, float x_f, float y_f, int iter_num)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int indx = i + j*n_y;
	if (i < n_x && j < n_y && K[indx] == 0 ){
		float Zx_prev = Zx[indx];
		float Zy_prev = Zy[indx];
		Zx[indx] = Zx_prev*Zx_prev - Zy_prev*Zy_prev + (dx*j+x_i);
		Zy[indx] = 2.0*Zx_prev*Zy_prev + (dy*i+y_i);
		if ( Zx[indx]*Zx[indx] +  Zy[indx]*Zy[indx] > (x_f - x_i + y_f - y_i))
			K[indx] =  iter_num%16;
	}
//	printf("length_sq = %f \n", Zx[indx]*Zx[indx] +  Zy[indx]*Zy[indx]);
}

int main(int argc, char const *argv[])
{
	start = clock();
	float dx = (x_f - x_i)/n_x;
	float dy = (y_f - y_i)/n_y;
	size_t size = n_x*n_x*sizeof(float);

	// dim of Block grid
	dim3 B(32, 32);
	dim3 G(n_x/32 +1 , n_y/32 +1);
	// allocate memory
	float *h_Zx = (float *)malloc(size);
	float *h_Zy = (float *)malloc(size);
	int *h_k = (int *)malloc(size);
	// allocate GPU memory
	float *d_Zx = NULL;
	float *d_Zy = NULL;
	int *d_k = NULL;
	hipMalloc((void **)&d_Zx, size);
	hipMalloc((void **)&d_Zy, size);
	hipMalloc((void **)&d_k, size);


	// init value
	for (unsigned int i=0; i<n_x; i++){
    	for (unsigned int j=0; j<n_y; j++){
    		h_Zx[i+j*n_y] = h_Zy[i+j*n_y] = h_k[i+j*n_y] = 0;
    	}
    }
    // move memory to device
//    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_Zx, h_Zx, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Zy, h_Zy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, size, hipMemcpyHostToDevice);

  //  printf("CUDA kernel launch with %dx%dx%d blocks of %dx%dx%d threads\n", B.x, B.y, B.z, G.x, G.y, G.z);

    for (unsigned int l=0; l < max_iter; l++){
    	MandelFunc<<<G, B>>>(d_Zx, d_Zy, d_k, dx, dy, x_i, y_i, x_f, y_f, l);
    }
    
    hipMemcpy(h_k, d_k, size, hipMemcpyDeviceToHost);

    file = fopen("mandel_gpu.ppm", "w");
	fprintf(file, "P2 %d %d 16\n", n_x, n_y);
	for (unsigned int i=0; i<n_x; i++){
		for (unsigned int j=0; j<n_y; j++){
	    		fprintf(file, "%d \t", h_k[i+j*n_y]);
		}
	    fprintf(file, "\n");
	}
	fprintf(file, "\n");
	fclose(file);

    // free device global memory
    hipFree(d_Zx);
    hipFree(d_Zy);
    hipFree(d_k);
    // free host memory
    free(h_Zx);
    free(h_Zy);
    free(h_k);

    end = clock();
    printf("GPU time = %lf\n", (double) (end - start)/ CLOCKS_PER_SEC );

	return 0;
}
