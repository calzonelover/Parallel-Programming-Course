#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "mandelbrot.h"

__global__ void 
MandelFunc (float *Zx, float *Zy, float *K
,int n_x, int n_y, float dx, float dy
, float x_i, float y_i, float x_f, float y_f, int iter_num)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < n_x && j < n_y){
		if ( Zx[indx]*Zx[indx] + Zy[indx]*Zy[indx] < (x_f - x_i + y_f - y_i)) {
			int indx = i + j*n_x;
			float Zx_prev = Zx[indx];
			float Zy_prev = Zy[indx];
			Zx[indx] = Zx_prev*Zx_prev - Zy_prev*Zy_prev + (dx*i+x_i);
			Zy[indx] = 2.0*Zx_prev*Zy_prev + (dy*j+y_i);
		} else {
			K[indx] = iter_num%16;
		}
	}
}

int main(int argc, char const *argv[])
{
	int data[n_x][n_y];
	float dx = (x_f - x_i)/n_x;
	float dy = (y_f - y_i)/n_y;
	int k;
	Complex z_prev;
	size_t size = n_x*n_x*sizeof(float)

	// dim of Block grid
	dim3 B(32, 32);
	dim3 G(n_x/32 +1 , n_y/32 +1);
	// allocate memory
	float *h_Zx = (float *)malloc(size);
	float *h_Zy = (float *)malloc(size);
	float *h_k = (float *)malloc(size);
	// allocate GPU memory
	float *d_Zx = NULL;
	float *d_Zy = NULL;
	float *d_k = NULL;
	hipMalloc((void **)&d_Zx, size);
	hipMalloc((void **)&d_Zy, size);
	hipMalloc((void **)&d_k, size);

	start = clock();

	// init value
	for (unsigned int i=0; i<n_x; i++){
    	for (unsigned int j=0; j<n_y; j++){
    		h_Zx[i+j*n_x] = h_Zy[i+j*n_y] = h_k[i+j*n_y] = 0;
    	}
    }
    // move memory to device
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_Zx, h_Zx, size, hipMemcpyDeviceToHost);
    hipMemcpy(d_Zy, h_Zy, size, hipMemcpyDeviceToHost);

    printf("CUDA kernel launch with %dx%dx%d blocks of %dx%dx%d threads\n", B.x, B.y, B.z, G.x, G.y, G.z);

    for (unsigned int l=0; l < max_iter; l++){
    	MandelFunc<<<G, B>>>(d_Zx, d_Zy, d_k, n_x, n_y, dx, dy, x_i, y_i, x_f, y_f, l);
    }
    
    end = clock();
    printf("GPU time = %lf\n", (double) (end - start)/ CLOCKS_PER_SEC );

    file = fopen("mandel.ppm", "w");
	fprintf(file, "P2 %d %d 16\n", n_x, n_y);
	for (unsigned int i=0; i<n_x; i++){
		for (unsigned int j=0; j<n_y; j++){
	    	fprintf(file, "%d \t", h_k[i+j*n_x]);
		}
	    fprintf(file, "\n");
	}
	fprintf(file, "\n");
	fclose(file);

    // free device global memory
    hipFree(d_Zx);
    hipFree(d_Zy);
    hipFree(d_k);
    // free host memory
    free(h_Zx);
    free(h_Zy);
    free(h_k);


	return 0;
}