#include "hip/hip_runtime.h"
#include "wave_cudaMpi.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void kernel_stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int global_iy = NY/2*_rank + iy;
    int indx = ix+iy*NX;
    if (ix > 0 && global_iy > 0 && ix < NX -1 && global_iy < NY - 1){
        if (_rank == 0 && global_iy == NY/2 -1){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _my_recv_halo[ix]+_wave2d_u1[ix+(iy-1)*NX]);
        } else if (_rank == 1 && global_iy == NY/2){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_my_recv_halo[ix]);
        } else {
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_wave2d_u1[ix+(iy-1)*NX]);
        }
    }
}

__global__ void kernel_updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1){
        int indx = ix+iy*nx;
        _wave2d_u0[indx] = _wave2d_u1[indx];
        _wave2d_u1[indx] = _wave2d_u2[indx];
    }
}

extern "C" void alloc_var_device(float *d_var, size_t size){
    hipMalloc((void **)&d_var, size);
}

extern "C" void inject_var_to_device(float *d_var, float *h_var, size_t size){
    hipMalloc((void **)&d_var, size);
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

extern "C" void load_var_to_host(float *h_var, float *d_var, size_t size){
    hipMemcpy(h_var, d_var, size, hipMemcpyDeviceToHost);
}

extern "C" void load_var_to_device(float *d_var, float *h_var, size_t size){
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

extern "C" void stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    dim3 B(32, 32);
    dim3 G(NX/32+1, NY/32+1);
    kernel_stepWave<<<G,B>>>(_wave2d_u0, _wave2d_u1, _wave2d_u2, my_recv_halo, _rank, _C2);
}

extern "C" void updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
    dim3 B(32, 32);
    dim3 G(NX/32+1, NY/32+1);
    kernel_stepWave<<<G,B>>>(_wave2d_u0, _wave2d_u1, _wave2d_u2, _rank);
}