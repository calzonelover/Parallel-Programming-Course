#include "hip/hip_runtime.h"
#include "wave_cudaMpi.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



__global__ void kernel_stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int global_iy = NY/2*_rank + iy;
    int indx = ix+iy*NX;
    if (ix > 0 && global_iy > 0 && ix < NX -1 && global_iy < NY - 1){
        if ( _rank == 0 && iy == NY/2 - 1 ){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*NX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _my_recv_halo[ix]+_wave2d_u1[ix+(iy-1)*NX]);
        } else if ( _rank == 1 && iy == 0 ){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*NX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_my_recv_halo[ix]);
        } else {
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*NX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_wave2d_u1[ix+(iy-1)*NX]);
        }
    }
}

__global__ void kernel_updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    if (ix > 0 && iy > 0 && ix < NX-1 && iy < NY-1){
        int indx = ix+iy*NX;
        _wave2d_u0[indx] = _wave2d_u1[indx];
        _wave2d_u1[indx] = _wave2d_u2[indx];
    }
}

extern "C" void alloc_var_device(float *d_var, size_t size){
    hipMalloc((void **)&d_var, size);
}

extern "C" void inject_var_to_device(float *d_var, float *h_var, size_t size){
    hipMalloc((void **)&d_var, size);
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

extern "C" void load_var_to_host(float *h_var, float *d_var, size_t size){
    hipMemcpy(h_var, d_var, size, hipMemcpyDeviceToHost);
}

extern "C" void load_var_to_device(float *d_var, float *h_var, size_t size){
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

extern "C" void stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    dim3 B(32, 32);
    dim3 G(NX/32+1, (NY/2)/32+1);
    size_t _my_size = NX*(NY/2)*sizeof(float);
    size_t _size_vec = NX*sizeof(float);

    float *_d_wave2d_u0 = NULL, *_d_wave2d_u1 = NULL, *_d_wave2d_u2 = NULL;
    hipMalloc((void **)&_d_wave2d_u0, _my_size);
    hipMalloc((void **)&_d_wave2d_u1, _my_size);
    hipMalloc((void **)&_d_wave2d_u2, _my_size);
    float *_d_my_recv_halo = NULL;

    hipMalloc((void **)&_d_my_recv_halo, _size_vec);

    hipMemcpy(_d_wave2d_u0, _wave2d_u0, _my_size, hipMemcpyHostToDevice);
    hipMemcpy(_d_wave2d_u1, _wave2d_u1, _my_size, hipMemcpyHostToDevice);
    hipMemcpy(_d_wave2d_u2, _wave2d_u2, _my_size, hipMemcpyHostToDevice);
    hipMemcpy(_d_my_recv_halo, _my_recv_halo, _size_vec, hipMemcpyHostToDevice);

    kernel_stepWave<<<G,B>>>(_d_wave2d_u0, _d_wave2d_u1, _d_wave2d_u2, _d_my_recv_halo, _rank, _C2);

    hipMemcpy(_wave2d_u0, _d_wave2d_u0, _my_size hipMemcpyDeviceToHost);
    hipMemcpy(_wave2d_u1, _d_wave2d_u1, _my_size hipMemcpyDeviceToHost);
    hipMemcpy(_wave2d_u2, _d_wave2d_u2, _my_size hipMemcpyDeviceToHost);

    hipFree(_d_wave2d_u0);
    hipFree(_d_wave2d_u1);
    hipFree(_d_wave2d_u2);
}

extern "C" void updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
    dim3 B(32, 32);
    dim3 G(NX/32+1, (NY/2)/32+1);
    size_t _my_size = NX*(NY/2)*sizeof(float);

    float *_d_wave2d_u0 = NULL, *_d_wave2d_u1 = NULL, *_d_wave2d_u2 = NULL;
    hipMalloc((void **)&_d_wave2d_u0, _my_size);
    hipMalloc((void **)&_d_wave2d_u1, _my_size);
    hipMalloc((void **)&_d_wave2d_u2, _my_size);

    hipMemcpy(_d_wave2d_u0, _wave2d_u0, _my_size, hipMemcpyHostToDevice);
    hipMemcpy(_d_wave2d_u1, _wave2d_u1, _my_size, hipMemcpyHostToDevice);
    hipMemcpy(_d_wave2d_u2, _wave2d_u2, _my_size, hipMemcpyHostToDevice);

    kernel_updateWave<<<G,B>>>(_d_wave2d_u0, _d_wave2d_u1, _d_wave2d_u2, _rank);

    hipMemcpy(_wave2d_u0, _d_wave2d_u0, _my_size hipMemcpyDeviceToHost);
    hipMemcpy(_wave2d_u1, _d_wave2d_u1, _my_size hipMemcpyDeviceToHost);
    hipMemcpy(_wave2d_u2, _d_wave2d_u2, _my_size hipMemcpyDeviceToHost);    

    hipFree(_d_wave2d_u0);
    hipFree(_d_wave2d_u1);
    hipFree(_d_wave2d_u2);
}