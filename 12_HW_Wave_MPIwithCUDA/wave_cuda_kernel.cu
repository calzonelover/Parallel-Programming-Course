#include "hip/hip_runtime.h"
#include "wave_cudaMPI.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


extern "C" void inject_var_to_device(float *h_var, float *d_var, size_t size){
    hipMalloc((void **)&d_var, size);
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

extern "C" void load_var_to_host(float *wave2d_u0, float *wave2d_u1, float *wave2d_u2, float *d_wave2d_u0, float *d_wave2d_u1, float *d_wave2d_u2, size_t my_size){
    hipMemcpy(wave2d_u0, d_wave2d_u0, my_size, hipMemcpyDeviceToHost);
    hipMemcpy(wave2d_u1, d_wave2d_u1, my_size, hipMemcpyDeviceToHost);
    hipMemcpy(wave2d_u2, d_wave2d_u2, my_size, hipMemcpyDeviceToHost);
}

extern "C" void load_var_to_device(float *h_var, float *d_var, size_t size){
    hipMemcpy(d_var, h_var, size, hipMemcpyHostToDevice);
}

__global__ void kernel_stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int global_iy = NY/2*_rank + iy;
    int indx = ix+iy*NX;
    if (ix > 0 && global_iy > 0 && ix < NX -1 && global_iy < NY - 1){
        if (_rank == 0 && global_iy == NY/2 -1){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _my_recv_halo[ix]+_wave2d_u1[ix+(iy-1)*NX]);
        } else if (_rank == 1 && global_iy == NY/2){
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_my_recv_halo[ix]);
        } else {
            _wave2d_u2[indx] = (2.0f-4.0f*_C2)*_wave2d_u1[indx] - _wave2d_u0[indx]
                        + _C2*(_wave2d_u1[(ix+1)+iy*nX]+_wave2d_u1[(ix-1)+iy*NX]
                        + _wave2d_u1[ix+(iy+1)*NX]+_wave2d_u1[ix+(iy-1)*NX]);
        }
    }
}

extern "C" void stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
    
}

__global__ void kernel_updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    if (ix > 0 && iy > 0 && ix < nx-1 && iy < ny-1){
        int indx = ix+iy*nx;
        _wave2d_u0[indx] = _wave2d_u1[indx];
        _wave2d_u1[indx] = _wave2d_u2[indx];
    }
}

extern "C" void updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){

}


// void stepWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, float *_my_recv_halo, int _rank, float _C2){
//     unsigned int _global_y_i;
//     for (unsigned int y_i = 0; y_i<NY/2; y_i++){
//         _global_y_i = NY/2*_rank + y_i;
//         // if (t_i == 2 && rank == 0) printf("rank %d y_i %d \n", rank, y_i);
//         for (unsigned int x_i=0; x_i<NX; x_i++){
//             if (x_i > 0 && _global_y_i > 0 && x_i < NX-1 && _global_y_i < NY-1){
//                 if (_rank == 0 && _global_y_i == NY/2-1){
//                 _wave2d_u2[y_i*NX+x_i] = (2.0f-4.0f*_C2)*_wave2d_u1[y_i*NX+x_i] - _wave2d_u0[y_i*NX+x_i]
//                                         + _C2*(_wave2d_u1[y_i*NX+(x_i+1)]+_wave2d_u1[y_i*NX+(x_i-1)]
//                                         + _my_recv_halo[x_i]+_wave2d_u1[(y_i-1)*NX+x_i]);
//                 } else if (_rank == 1 && _global_y_i == NY/2){
//                 _wave2d_u2[y_i*NX+x_i] = (2.0f-4.0f*_C2)*_wave2d_u1[y_i*NX+x_i] - _wave2d_u0[y_i*NX+x_i]
//                                         + _C2*(_wave2d_u1[y_i*NX+(x_i+1)]+_wave2d_u1[y_i*NX+(x_i-1)]
//                                         + _wave2d_u1[(y_i+1)*NX+x_i]+_my_recv_halo[x_i]);
//                 } else {
//                 _wave2d_u2[y_i*NX+x_i] = (2.0f-4.0f*_C2)*_wave2d_u1[y_i*NX+x_i] - _wave2d_u0[y_i*NX+x_i]
//                                         + _C2*(_wave2d_u1[y_i*NX+(x_i+1)]+_wave2d_u1[y_i*NX+(x_i-1)]
//                                         + _wave2d_u1[(y_i+1)*NX+x_i]+_wave2d_u1[(y_i-1)*NX+x_i]);
//                 }
//             }
//         }
//     }
// }

// void updateWave(float *_wave2d_u0, float *_wave2d_u1, float *_wave2d_u2, int _rank){
//     unsigned int _global_y_i;
//     for (unsigned int y_i = 0; y_i<NY/2; y_i++){
//         _global_y_i = NY/2*_rank + y_i;
//         for (unsigned int x_i=0; x_i<NX; x_i++){
//             if (x_i > 0 && _global_y_i > 0 && x_i < NX-1 && _global_y_i < NY-1){
//                 _wave2d_u0[y_i*NX+x_i] = _wave2d_u1[y_i*NX+x_i];
//                 _wave2d_u1[y_i*NX+x_i] = _wave2d_u2[y_i*NX+x_i];
//             }
//         }
//     }
// }